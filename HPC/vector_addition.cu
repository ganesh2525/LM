#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
using namespace std;

// Error checking macro
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        cerr << "CUDA Error: " << hipGetErrorString(code) << " " << file << " " << line << endl;
        exit(code);
    }
}

// Sequential vector addition
void vectorAddSequential(int *a, int *b, int *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for parallel vector addition
__global__ void vectorAddParallel(int *a, int *b, int *c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 1 << 24;
    size_t size = n * sizeof(int);

    // Host allocations
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);
    int *h_c = (int*)malloc(size);
    int *h_c_parallel = (int*)malloc(size);

    // Initialize host vectors
    srand(time(0));
    for (int i = 0; i < n; i++) {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    // Device allocations
    int *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, size));
    CUDA_CHECK(hipMalloc(&d_b, size));
    CUDA_CHECK(hipMalloc(&d_c, size));

    // Copy to device
    CUDA_CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    // Sequential execution
    clock_t start_time = clock();
    vectorAddSequential(h_a, h_b, h_c, n);
    clock_t end_time = clock();
    double seq_time = double(end_time - start_time) / CLOCKS_PER_SEC;
    cout << "Sequential vector addition time: " << seq_time << " seconds" << endl;

    // CUDA execution
    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    start_time = clock();
    vectorAddParallel<<<blocks, threads>>>(d_a, d_b, d_c, n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    end_time = clock();
    double parallel_time = double(end_time - start_time) / CLOCKS_PER_SEC;
    cout << "Parallel vector addition time: " << parallel_time << " seconds" << endl;

    // Copy result to host
    CUDA_CHECK(hipMemcpy(h_c_parallel, d_c, size, hipMemcpyDeviceToHost));

    // Output sample results
    // cout << "\nIndex\t\th_a\th_b\t\th_c (Seq)\th_c_parallel (CUDA)" << endl;
    // for (int i = 0; i < n; i++) {
    //     cout << i << "\t\t"
    //          << h_a[i] << "\t"
    //          << h_b[i] << "\t\t"
    //          << h_c[i] << "\t\t"
    //          << h_c_parallel[i] << endl;
    // }

    // Free memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_parallel);

    return 0;
}


// %%writefile vector_addition.cu
// !nvcc -arch=sm_75 vector_addition.cu -o vector_addition
// !./vector_addition